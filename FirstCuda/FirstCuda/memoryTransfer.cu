#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <vector>
#include <memory>

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char * file, int line, bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

// Device Code
__global__ void mem_transfer_test(int* input, int size) {

	int tid = threadIdx.x + threadIdx.y * blockDim.x;
	int block_offset = blockIdx.x * (blockDim.x * blockDim.y);
	int row_offset = blockIdx.y * (gridDim.x * blockDim.x * blockDim.y);
	int gid = tid + block_offset + row_offset;
	if (gid < size) {
		printf("tid: %d, gid: %d, value: %d \n", tid, gid, input[gid]);
	}
}

// Host Code
int mainMem() {

	int array_size = 150;
	int array_byte_size = sizeof(int) * array_size;
	std::unique_ptr<int[]> host_input = std::make_unique<int[]>(array_size);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < array_size; i++) {
		host_input[i] = static_cast<int>(rand() & 0xff);
	}

	int* device_input = nullptr;

	hipError_t err;

	// Allocate memory on GPU
	err = hipMalloc((void**)&device_input,array_byte_size);	// To allocate memory on GPU
	if (err != hipSuccess) {
		std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// Alternative with Macro
	gpuErrorCheck(hipMemcpy(device_input, host_input.get(), array_byte_size, hipMemcpyHostToDevice));


	dim3 block(32, 1, 1);
	dim3 grid(5, 1, 1);
	mem_transfer_test << < grid, block >> > (device_input, array_size);	// Asynchronous function call

	// We we need to wait for the kernel function to be done we need to use
	err = hipDeviceSynchronize();	// Similar to .join() in std::thread but it is for all kernels that were launched
	if (err != hipSuccess) {
		std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(err) << std::endl;
		hipFree(device_input);
		return -1;
	}

	hipFree(device_input);
	hipDeviceReset();

	return 0;
}