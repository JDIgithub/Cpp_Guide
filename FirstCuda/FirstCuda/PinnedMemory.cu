#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

#include <vector>
#include <memory>



int main() {

	// Memory Size 128 MBs
	int isize = 1 << 25;
	int nbytes = isize * sizeof(float);

	// Allocate the Host Memory
	float * host_a = new float[isize];
	// Allocate the Pinned Memory
	float * host_a_pinned;
	hipHostMalloc((float **)&host_a_pinned, nbytes, hipHostMallocDefault);

	// Allocate the Device Memory
	float * device_a = nullptr;
	hipMalloc((float **)&device_a, nbytes);

	// Init the Host memory
	for (int i = 0; i < isize; i++) host_a[i] = 7;

	// Transfer data from the host to device
	hipMemcpy(device_a, host_a, nbytes, hipMemcpyHostToDevice);
	// Transfer  back from device to host
	hipMemcpy(host_a, device_a, nbytes, hipMemcpyDeviceToHost);

	// The same with Pinned memory
	// These transfers should be faster
	hipMemcpy(device_a, host_a_pinned, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(host_a_pinned, device_a, nbytes, hipMemcpyDeviceToHost);

	// free Memory
	delete[] host_a;
	hipHostFree(host_a_pinned);
	hipFree(device_a);
	
	hipDeviceReset();

	return 0;
}